#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <time.h>
#include <float.h>
#include <hiprand/hiprand_kernel.h>
#include "vec3.h"
#include "ray.h"
#include "sphere.h"
#include "hitable_list.h"
#include "camera.h"
#include "material.h"
#include "texture.h"

#define STB_IMAGE_IMPLEMENTATION
#include "stb/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb/stb_image_write.h"
#define STB_IMAGE_RESIZE_IMPLEMENTATION
#include "stb/stb_image_resize.h"

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

// Matching the C++ code would recurse enough into color() calls that
// it was blowing up the stack, so we have to turn this into a
// limited-depth loop instead.  Later code in the book limits to a max
// depth of 50, so we adapt this a few chapters early on the GPU.
__device__ vec3 color(const ray& r, hitable** world, hiprandState* local_rand_state) {
    ray cur_ray = r;
    vec3 cur_attenuation = vec3(1.0, 1.0, 1.0);
    for (int i = 0; i < 50; i++) {
        hit_record rec;
        if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
            ray scattered;
            vec3 attenuation;
            if (rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered, local_rand_state)) {
                cur_attenuation *= attenuation;
                cur_ray = scattered;
            }
            else {
                return vec3(0.0, 0.0, 0.0);
            }
        }
        else {
            vec3 unit_direction = unit_vector(cur_ray.direction());
            float t = 0.5f * (unit_direction.y() + 1.0f);
            vec3 c = (1.0f - t) * vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.7, 1.0);
            return cur_attenuation * c;
        }
    }
    return vec3(0.0, 0.0, 0.0); // exceeded recursion
}

__global__ void rand_init(hiprandState* rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprand_init(1984, 0, 0, rand_state);
    }
}

__global__ void render_init(int max_x, int max_y, hiprandState* rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    // Original: Each thread gets same seed, a different sequence number, no offset
    // hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
    // BUGFIX, see Issue#2: Each thread gets different seed, same sequence for
    // performance improvement of about 2x!
    hiprand_init(1984 + pixel_index, 0, 0, &rand_state[pixel_index]);
}

__global__ void render(vec3* fb, int max_x, int max_y, int ns, camera** cam, hitable** world, hiprandState* rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    hiprandState local_rand_state = rand_state[pixel_index];
    vec3 col(0, 0, 0);
    for (int s = 0; s < ns; s++) {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = (*cam)->get_ray(u, v, &local_rand_state);
        col += color(r, world, &local_rand_state);
    }
    rand_state[pixel_index] = local_rand_state;
    col /= float(ns);
    col[0] = sqrt(col[0]);
    col[1] = sqrt(col[1]);
    col[2] = sqrt(col[2]);
    fb[pixel_index] = col;
}

#define RND (hiprand_uniform(&local_rand_state))

__global__ void create_world(hitable** d_list, hitable** d_world, camera** d_camera, int nx, int ny, hiprandState* rand_state, unsigned char* dataPtr) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        Texture* image = new ImageTexture(dataPtr, 128, 128);

        hiprandState local_rand_state = *rand_state;
        Texture* checker = new CheckerTexture(new ConstantTexture(vec3(0.2, 0.3, 0.1)), new ConstantTexture(vec3(0.9, 0.9, 0.9)));
        Texture* color = new ConstantTexture(vec3(128.0 / 255, 0, 128.0 / 255));
        d_list[0] = new sphere(vec3(0, -1000.0, -1), 1000, new lambertian(checker)); 
        int i = 1;
        d_list[i++] = new sphere(vec3(0, 1, 0), 1.0, new dielectric(1.5));
        d_list[i++] = new sphere(vec3(-4, 1, 0), 1.0, new lambertian(color));
        d_list[i++] = new sphere(vec3(4, 1, 0), 1.0, new metal(vec3(0.7, 0.6, 0.5), 0.0));
        d_list[i++] = new sphere(vec3(4, 1.0, 2), 1.0, new lambertian(image));
        *rand_state = local_rand_state;
        *d_world = new hitable_list(d_list, 1 + 4);

        vec3 lookfrom(13, 6, 6);
        vec3 lookat(0, 0, 0);
        float dist_to_focus = 10.0; (lookfrom - lookat).length();
        float aperture = 0.1;
        *d_camera = new camera(lookfrom,
            lookat,
            vec3(0, 1, 0),
            30.0,
            float(nx) / float(ny),
            aperture,
            dist_to_focus);
    }
}

__global__ void free_world(hitable** d_list, hitable** d_world, camera** d_camera) {
    for (int i = 0; i < 1 + 4; i++) {
        delete ((sphere*)d_list[i])->mat_ptr;
        delete d_list[i];
    }
    delete* d_world;
    delete* d_camera;
}

int main() {
    int nx = 1200;
    int ny = 800;
    int ns = 10;
    int tx = 8;
    int ty = 8;

    
    std::cerr << "Rendering a " << nx << "x" << ny << " image with " << ns << " samples per pixel ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = nx * ny;
    size_t fb_size = num_pixels * sizeof(vec3);

    // allocate FB
    vec3* fb;
    checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

    // allocate random state
    hiprandState* d_rand_state;
    checkCudaErrors(hipMalloc((void**)&d_rand_state, num_pixels * sizeof(hiprandState)));
    hiprandState* d_rand_state2;
    checkCudaErrors(hipMalloc((void**)&d_rand_state2, 1 * sizeof(hiprandState)));

    // we need that 2nd random state to be initialized for the world creation
    rand_init << <1, 1 >> > (d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // make our world of hitables & the camera
    int iw, ih, n;
    unsigned char* idata = stbi_load("logo.jpg", &iw, &ih, &n, 0);
    int ow = 128;
    int oh = 128;
    auto* odata = (unsigned char*)malloc(ow * oh * n);
    stbir_resize(idata, iw, ih, 0, odata, ow, oh, 0, STBIR_TYPE_UINT8, n, STBIR_ALPHA_CHANNEL_NONE, 0,
        STBIR_EDGE_CLAMP, STBIR_EDGE_CLAMP,
        STBIR_FILTER_BOX, STBIR_FILTER_BOX,
        STBIR_COLORSPACE_SRGB, nullptr
    );
    unsigned char* dataPtr;
    checkCudaErrors(hipMalloc((void**)&dataPtr, ow * oh * 3 * sizeof(unsigned char)));
    checkCudaErrors(hipMemcpy(dataPtr, odata, oh * ow * 3 * sizeof(unsigned char), hipMemcpyHostToDevice));

    hitable** d_list;
    int num_hitables = 1 + 4;
    checkCudaErrors(hipMalloc((void**)&d_list, num_hitables * sizeof(hitable*)));
    hitable** d_world;
    checkCudaErrors(hipMalloc((void**)&d_world, sizeof(hitable*)));
    camera** d_camera;
    checkCudaErrors(hipMalloc((void**)&d_camera, sizeof(camera*)));
    create_world << <1, 1 >> > (d_list, d_world, d_camera, nx, ny, d_rand_state2, dataPtr);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    clock_t start, stop;
    start = clock();
    // Render our buffer
    dim3 blocks(nx / tx + 1, ny / ty + 1);
    dim3 threads(tx, ty);
    render_init << <blocks, threads >> > (nx, ny, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    std::cerr << "render init\n";
    render << <blocks, threads >> > (fb, nx, ny, ns, d_camera, d_world, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    std::cerr << "render done\n";
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";

    // Output FB as Image
    std::cout << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny - 1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j * nx + i;
            int ir = int(255.99 * fb[pixel_index].r());
            int ig = int(255.99 * fb[pixel_index].g());
            int ib = int(255.99 * fb[pixel_index].b());
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }

    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    free_world << <1, 1 >> > (d_list, d_world, d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(d_rand_state2));
    checkCudaErrors(hipFree(fb));

    hipDeviceReset();
}
